#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float x = input[idx];
        output[idx] = x * (x < 0.0f ? 0.01f : 1.0f);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}

