#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <hip/hip_runtime.h>

#define TILE_DIM 16

__global__ void matrix_transpose_kernel(const float* input, 
                                        float* output, 
                                        int rows, 
                                        int cols) {
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    // Original IDs
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    // Save elements into tile
    if (x < cols && y < rows) {
        tile[threadIdx.y][threadIdx.x] = input[y * cols + x];
    }

    __syncthreads();

    // Transpose IDs
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    if (x < rows && y < cols) {
        output[y * rows + x] = tile[threadIdx.x][threadIdx.y];
    }
}


// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}
