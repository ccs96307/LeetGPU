#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < width * height) {
        int base = idx * 4;
        image[base] = 255 - image[base];
        image[base+1] = 255 - image[base+1];
        image[base+2] = 255 - image[base+2];
    }
}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}

